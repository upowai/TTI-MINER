#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include "sha256.cuh"

__global__ void compute_sha256(const BYTE *input, size_t length, BYTE *output)
{
    SHA256_CTX ctx;
    sha256_init(&ctx);
    sha256_update(&ctx, input, length);
    sha256_final(&ctx, output);

    // Debug print inside the kernel
}

extern "C" void hash_string(const char *str, char *hash_output)
{
    BYTE *dev_input, *dev_output;
    size_t length = strlen(str);
    BYTE host_output[SHA256_BLOCK_SIZE];

    checkCudaErrors(hipMalloc((void **)&dev_input, length * sizeof(BYTE)));
    checkCudaErrors(hipMalloc((void **)&dev_output, SHA256_BLOCK_SIZE * sizeof(BYTE)));

    checkCudaErrors(hipMemcpy(dev_input, str, length * sizeof(BYTE), hipMemcpyHostToDevice));

    // Initialize the constant memory
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k)));

    compute_sha256<<<256, 256>>>(dev_input, length, dev_output);

    checkCudaErrors(hipMemcpy(host_output, dev_output, SHA256_BLOCK_SIZE * sizeof(BYTE), hipMemcpyDeviceToHost));

    hipFree(dev_input);
    hipFree(dev_output);

    char *result = hash_to_string(host_output);
    strcpy(hash_output, result);
    free(result);
}